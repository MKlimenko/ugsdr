#include "hip/hip_runtime.h"
#include "cuda_mixer.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <thrust/complex.h>
#include <thrust/device_vector.h>
#include <cmath>

namespace {
	template <typename T>
	__global__ void mul_kernel(T* src_dst, double relative_frequency, double phase) {
		int i = threadIdx.x + blockIdx.x * blockDim.x;
		double triarg = 2 * 4 * std::atan(1.0) * relative_frequency * i;
		src_dst[i] *= cos(triarg);
	}

	template <typename T>
	__global__ void mul_kernel(thrust::complex<T>* src_dst, double relative_frequency, double phase) {
		int i = threadIdx.x + blockIdx.x * blockDim.x;
		double triarg = 2 * 4 * std::atan(1.0) * relative_frequency * i;
		src_dst[i] *= thrust::exp(thrust::complex<T>(0, triarg));
	}

	template <typename T>
	void ProcessDevice(thrust::device_vector<T>& src_dst, double relative_frequency, double phase) {
		hipDeviceProp_t properties;
		hipSetDevice(0);
		hipGetDeviceProperties(&properties, 0);
		unsigned blocks_cnt = 0;
		unsigned threads_cnt = 0;

		for (std::size_t i = 1; i < src_dst.size(); ++i) {
			auto div_result = std::div(static_cast<std::ptrdiff_t>(src_dst.size()), i);
			if (div_result.rem)
				continue;
			if (div_result.quot <= properties.maxThreadsPerBlock) {
				blocks_cnt = static_cast<unsigned>(i);
				threads_cnt = static_cast<unsigned>(div_result.quot);
				break;
			}
		}

		mul_kernel<<<blocks_cnt, threads_cnt>>>(thrust::raw_pointer_cast(src_dst.data()), relative_frequency, phase);
	}
	template <typename T>
	void ProcessHost(std::vector<T>& src_dst, double relative_frequency, double phase) {
		thrust::device_vector<T> gpu(src_dst.begin(), src_dst.end());
		ProcessDevice(gpu, relative_frequency, phase);
		auto cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
			throw std::runtime_error("Oopsie");
		thrust::copy(gpu.begin(), gpu.end(), src_dst.begin());
	}
	template <typename T>
	void ProcessHost(std::vector<std::complex<T>>& src_dst, double relative_frequency, double phase) {
		thrust::device_vector<thrust::complex<T>>gpu(src_dst.begin(), src_dst.end());
		ProcessDevice(gpu, relative_frequency, phase);
		auto cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess)
			throw std::runtime_error("Oopsie");
		thrust::copy(gpu.begin(), gpu.end(), src_dst.begin());
	}
}

namespace ugsdr {
	template <>
	void CudaMixer::Process<float>(std::vector<std::complex<float>>& src_dst, double sampling_freq, double frequency, double phase) {
		ProcessHost(src_dst, frequency / sampling_freq, phase);
	}
	template <>
	void CudaMixer::Process<double>(std::vector<std::complex<double>>& src_dst, double sampling_freq, double frequency, double phase) {
		ProcessHost(src_dst, frequency / sampling_freq, phase);
	}
}
